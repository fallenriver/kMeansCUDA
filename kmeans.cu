#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2009 by Miltiadis Allamanis
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy
 *  of this software and associated documentation files (the "Software"), to deal
 *  in the Software without restriction, including without limitation the rights
 *  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *  copies of the Software, and to permit persons to whom the Software is
 *  furnished to do so, subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in
 *  all copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 *  THE SOFTWARE.
 */

/**
 * K-means in Thrust
 *
 */
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/transform.h>
#include <thrust/segmented_scan.h>
#include <thrust/gather.h>
#include <thrust/scatter.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/tuple.h>
#include <thrust/utility.h>
#include <thrust/fill.h>
#include <thrust/adjacent_difference.h>
#include <thrust/remove.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>

#include <thrust/sorting/radix_sort.h>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <cstdlib>
#include <stdlib.h>
#include <math.h>

#define BUF_SIZE 1024

typedef thrust::tuple<float,int> IteratorPair;

template <typename T>
struct square
{
    __host__ __device__
        T operator()(const T& x) const {
            return x*x;
        }
};

template <typename T>
struct incr
{
    __host__ __device__
        T operator()(const T& x) const {
            return x+1;
        }
};

template <typename T>
struct unitify
{
    __host__ __device__
        T operator()(const T& x) const {
            return x!=0?1:0;
        }
};

struct minPair{
    __host__ __device__
        IteratorPair operator ()(const IteratorPair& a, const IteratorPair& b) const
        {
            if (thrust::get<0>(a) < thrust::get<0>(b))
                return a;
            else
                return b;
        }
};

struct division{
    __host__ __device__
        float operator ()(const float& a, const int& b) const
        {
            if (b==0) return a;
            else return a/b;
        }
};

struct sqDiff{
    __host__ __device__
        float operator ()(const float& a, const int& b) const
        {
            return (a-b)*(a-b);
        }
};

int main(int argc, char** argv)
{
    //Check for wrong parameter count
    if (argc!=3){
        std::cout<<"Error: Wrong number of parameters\n";
        exit(1);
    }

    /*------------------------------------------------------------------
     *  Read input file and store
     *----------------------------------------------------------------*/

    std::cout<<"Loading points from "<<argv[1]<<"...\n";
    unsigned int numAttributes=0,numObjects=0;;
    char line[BUF_SIZE]; //Temporary buffer
    FILE *infile;

    //Open file if possible
    if ((infile = fopen(argv[1], "r")) == NULL) {
            fprintf(stderr, "Error: no such file (%s)\n", argv[1]);
            exit(1);
    }

    while (fgets(line, BUF_SIZE, infile) != NULL)
            if (strtok(line, " \t\n") != 0)
                numObjects++;

    rewind(infile);
    while (fgets(line, BUF_SIZE, infile) != NULL) {
        if (strtok(line, " \t\n") != 0) {
            /* ignore the id (first attribute): numAttributes = 1; */
            while (strtok(NULL, " ,\t\n") != NULL) numAttributes++;
                break;
        }
   }

    rewind(infile); //Goto the beggining

    thrust::host_vector<float> feature(numObjects*numAttributes);

    int i=0;
     while (fgets(line, BUF_SIZE, infile) != NULL) {
            if (strtok(line, " \t\n") == NULL) continue;
            for (unsigned int j=0; j<numAttributes; j++) {
                feature[i]=atof(strtok(NULL, " ,\t\n"));

                i++;
          }
     }

    fclose(infile);  //Close File

    /************************END OF FILE LOADING***********************/


    //Setting number of clusters
    std::cout<<"Using "<<argv[2]<<" clusters.\n";
    int numClusters=atoi(argv[2]);
    thrust::host_vector<float> clusters(numClusters*numAttributes);
    srand(time(NULL));
    //Random Init Clusters
    for (int i=0;i<numClusters;i++){
        int randIndex=rand() % numObjects;
        std::cout<<"Random cluster selected: "<<randIndex<<"\n";
        for (unsigned int j=0;j<numAttributes;j++)
            clusters[i*numAttributes+j]=feature[randIndex*numAttributes+j];
    }

    /*------------------------------------------------------------------
     *  Start k-means
     * ---------------------------------------------------------------*/

    // Create device vectors and transfer to device
    thrust::device_vector<float> dFeature = feature;
    thrust::device_vector<float> dClusters = clusters;

    thrust::device_vector<int> dNumObjects(1);
    dNumObjects[0]=numObjects;

    thrust::device_vector<int> dNumAttributes(1);
    dNumAttributes[0]=(int)numAttributes;

    //Temporary float Vectors
    thrust::device_vector<float> dTemporary2(numObjects);
    thrust::device_vector<float> dTemporary(numObjects*numAttributes);

    thrust::device_vector<float> dResults(numObjects*numClusters);

    //Create Masks and Maps
    thrust::counting_iterator<int> objCount(0);
    thrust::constant_iterator<int> attr(numAttributes);
    thrust::device_vector<int> segScanMask(numObjects*numAttributes);
    thrust::transform(objCount,objCount+numObjects*numAttributes,attr,segScanMask.begin(),thrust::divides<int>());

    thrust::device_vector<int> clusterMask(numObjects*numAttributes);
    thrust::transform(objCount,objCount+numObjects*numAttributes,attr,clusterMask.begin(),thrust::modulus<int>());

    thrust::device_vector<int> gatherDistMask(numObjects);
    thrust::sequence(gatherDistMask.begin(),gatherDistMask.end(),numAttributes-1,numAttributes);

    thrust::device_vector<int> clusterPointsMask(numObjects*numClusters);
    thrust::constant_iterator<int> clust(numClusters);
    thrust::transform(objCount,objCount+numObjects*numClusters,clust,clusterPointsMask.begin(),thrust::modulus<int>());

    thrust::device_vector<int> pointMinMask(numObjects*numClusters);
    thrust::transform(objCount,objCount+numObjects*numClusters,clust,pointMinMask.begin(),thrust::divides<int>());

    thrust::device_vector<int> clusterCoordinate(numClusters*numAttributes);
    thrust::transform(objCount,objCount+numClusters*numAttributes,attr,clusterCoordinate.begin(),thrust::divides<int>());

    thrust::device_vector<int> scatterResults(dNumObjects[0]);

    thrust::device_vector<float> temporaryStorage(dNumObjects[0]);
    thrust::device_vector<float> temporaryStorage2(numClusters);

    thrust::device_vector<int> clusterMapping(numClusters);

    thrust::device_vector<int> divisionBy(numClusters*numAttributes);
    thrust::device_vector<int> objectCount(dNumObjects[0]);
    thrust::device_vector<int> clusterObjectCount(numClusters);


   for (unsigned int iterations=0;iterations<30;iterations++){
        std::cout<<"Iteration "<<iterations<<"...\n";//Start iteration

        thrust::sequence(scatterResults.begin(),scatterResults.end(),0,numClusters);//Caluclate scatterResult

        //For each cluster calculate distances
        for (int i=0;i<numClusters;i++){

            thrust::gather(dTemporary.begin(),dTemporary.end(),clusterMask.begin(),dClusters.begin()+i*dNumAttributes[0]); //Cluster center coordinates
            thrust::transform(dFeature.begin(),dFeature.end(),dTemporary.begin(),dTemporary.begin(),sqDiff()); //Calculate squares

            //Reduce sum per cluster point, using segmented scan...
            thrust::experimental::inclusive_segmented_scan(dTemporary.begin(),dTemporary.end(),segScanMask.begin(),dTemporary.begin())  ;

            thrust::gather(dTemporary2.begin(),dTemporary2.end(),gatherDistMask.begin(),dTemporary.begin()); //Gather all point distances to cluster
            thrust::scatter(dTemporary2.begin(),dTemporary2.end(),scatterResults.begin(),dResults.begin()); //Scatter result to result table
            thrust::transform(scatterResults.begin(),scatterResults.end(),thrust::make_constant_iterator(1),scatterResults.begin(),thrust::plus<int>()); //Calculate next scatterResults


        }//next cluster

        //Find best cluster for each point
            //SegScan for min pair...
            thrust::experimental::inclusive_segmented_scan( thrust::make_zip_iterator(make_tuple(dResults.begin(),clusterPointsMask.begin())),
                                                            thrust::make_zip_iterator(make_tuple(dResults.end(),clusterPointsMask.end())),
                                                            pointMinMask.begin(),
                                                            thrust::make_zip_iterator(make_tuple(dResults.begin(),clusterPointsMask.begin())),
                                                            minPair());

            //Gather clusterNumbers & points...

            thrust::device_vector<int> mapObjects(dNumObjects[0]);
            thrust::sequence(mapObjects.begin(),mapObjects.end(),numClusters-1,numClusters);

            thrust::device_vector<float> pointClusters(dNumObjects[0]);
            thrust::gather(pointClusters.begin(),pointClusters.end(),mapObjects.begin(),clusterPointsMask.begin());

            //Sort by cluster
            thrust::sequence(mapObjects.begin(),mapObjects.end(),0,(int)dNumAttributes[0]); //Create map from point numbers to point coordinate
            thrust::sorting::radix_sort_by_key(pointClusters.begin(),pointClusters.end(),mapObjects.begin());

            //Count points in cluster
                //Segmented Scan
                thrust::fill(objectCount.begin(),objectCount.end(),1);
                thrust::experimental::inclusive_segmented_scan(objectCount.begin(),objectCount.end(),pointClusters.begin(),objectCount.begin());

                //Gather to a vector

                //but first create the gather map
                thrust::device_vector<int> gatherSumsVector(dNumObjects[0]);

                thrust::adjacent_difference(pointClusters.begin(),pointClusters.end(),gatherSumsVector.begin());
                thrust::transform(gatherSumsVector.begin(),gatherSumsVector.end(),gatherSumsVector.begin(),unitify<int>());
                thrust::device_vector<int> activeClusters(1);
                activeClusters[0]=thrust::reduce(gatherSumsVector.begin(),gatherSumsVector.end())+1;
                thrust::transform(gatherSumsVector.begin(),gatherSumsVector.end(),thrust::make_counting_iterator(1),gatherSumsVector.begin(),thrust::multiplies<int>()); //This does not allow clusters with no points...
                thrust::remove(gatherSumsVector.begin(),gatherSumsVector.end(),0);
                //std::cout<<"Active Clusters: "<<activeClusters[0]<<"\n";

                thrust::transform(gatherSumsVector.begin(),gatherSumsVector.end(),thrust::make_constant_iterator(2),gatherSumsVector.begin(),thrust::minus<int>());
                gatherSumsVector[activeClusters[0]-1]=dNumObjects[0]-1; //Last sum will be at the last position

                thrust::device_vector<int> scatterClustersVector(activeClusters[0]);

                thrust::gather(scatterClustersVector.begin(),scatterClustersVector.end(),gatherSumsVector.begin(),pointClusters.begin());

                thrust::device_vector<int> clusterObjectCount(activeClusters[0]);
                thrust::gather(clusterObjectCount.begin(),clusterObjectCount.end(),gatherSumsVector.begin(),objectCount.begin());

            //Create map from point numbers to point coordinate
            //thrust::transform(mapObjects.begin(),mapObjects.end(),thrust::make_constant_iterator(numAttributes),mapObjects.begin(),thrust::multiplies<int>());

            //Create map from cluster coordinate to dClusters
            thrust::transform(scatterClustersVector.begin(),scatterClustersVector.end(),thrust::make_constant_iterator(numAttributes),scatterClustersVector.begin(),thrust::multiplies<int>());

            std::cout<<"Updating clusters...";
            for (unsigned int attrNo=0;attrNo<numAttributes;attrNo++){ //for each attribute (ie coordinate)
                    //gather all point coordinates
                    thrust::gather(temporaryStorage.begin(),temporaryStorage.end(),mapObjects.begin(),dFeature.begin());
                    //segmented scan (sum)
                    thrust::experimental::inclusive_segmented_scan(temporaryStorage.begin(),temporaryStorage.end(),pointClusters.begin(),temporaryStorage.begin());
                    //gather at cluster coordinates
                    thrust::gather(temporaryStorage2.begin(),temporaryStorage2.begin()+activeClusters[0],gatherSumsVector.begin(),temporaryStorage.begin());
                    //scatter to clusters
                    thrust::scatter(temporaryStorage2.begin(),temporaryStorage2.begin()+activeClusters[0],scatterClustersVector.begin(),dClusters.begin()); //Change
                    //increment map to point to next attribute
                    thrust::transform(mapObjects.begin(),mapObjects.end(),thrust::make_constant_iterator(1),mapObjects.begin(),thrust::plus<int>());
                    thrust::transform(scatterClustersVector.begin(),scatterClustersVector.end(),thrust::make_constant_iterator(1),scatterClustersVector.begin(),thrust::plus<int>());
            }

            //for each cluster divide with count by first scattering "count" and then divide-transform.
            thrust::gather(divisionBy.begin(),divisionBy.end(),clusterCoordinate.begin(),clusterObjectCount.begin());
            thrust::transform(dClusters.begin(),dClusters.end(),divisionBy.begin(),dClusters.begin(),division());
    }//next interation


    // Print output
  for (int g=0;g<numClusters;g++){
            std::cout<<"Cluster "<<g<<":(";
            for (int f=0;f<numAttributes;f++)
                std::cout<<dClusters[g*numAttributes+f]<<",";
            std::cout<<")\n";
  }
    return 0;
}



